#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kde.cuh"
#include "cudaErrorCheck.cu"

#define BLOCKSIZE 16

__global__ void kdeKernel(float * dPop, float * dCase, int nRow, int nCol, float xMin, float yMax, float cellSize, float * dX, float * dY, float * dP, float * dC, int * dPoints, float bandwidth2, int blockBandwidth)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int idInThread = threadIdx.y * blockDim.x + threadIdx.x;
 
	float cellX = xMin + cellSize * (j + 0.5);
	float cellY = yMax - cellSize * (i + 0.5);

	float denPop = 0.0f;
	float denCase = 0.0f;
	float dist2;
	float weight;

	int pointProcessed;
	int pointToProcess;
	int endPoint;

	__shared__ float sX[BLOCKSIZE*BLOCKSIZE];
	__shared__ float sY[BLOCKSIZE*BLOCKSIZE];
	__shared__ float sP[BLOCKSIZE*BLOCKSIZE];
	__shared__ float sC[BLOCKSIZE*BLOCKSIZE];


	for(int k = 0; k < 1 + 2 * blockBandwidth; k ++)
	{
		int dataBID = (blockIdx.y + k) * (gridDim.x + 2 * blockBandwidth)+ blockIdx.x;
		if(dataBID < 1)
		{
			pointProcessed = 0;
		}	
		else
		{
			pointProcessed = dPoints[dataBID - 1];
		}
		endPoint = dPoints[dataBID + 2 * blockBandwidth];

		pointToProcess = BLOCKSIZE * BLOCKSIZE;
	
		for(; pointProcessed < endPoint; pointProcessed += BLOCKSIZE * BLOCKSIZE)
		{
			if(pointProcessed + pointToProcess > endPoint)
			{	
				pointToProcess = endPoint - pointProcessed;
			}

			if(idInThread < pointToProcess)
			{
				sX[idInThread] = dX[pointProcessed + idInThread];
				sY[idInThread] = dY[pointProcessed + idInThread];
				sP[idInThread] = dP[pointProcessed + idInThread];
				sC[idInThread] = dC[pointProcessed + idInThread];
			}
			__syncthreads();
	

			for(int m = 0; m < pointToProcess; m++)
			{
				dist2 = (cellX - sX[m]) * (cellX - sX[m]) + (cellY - sY[m]) * (cellY - sY[m]);
				if(dist2 < bandwidth2)
				{
					weight = (1 - dist2 / bandwidth2);
					denPop += weight * sP[m];
					denCase += weight * sC[m];
				}
			}
				
			__syncthreads();
		}
	}
	
	if(i < nRow && j < nCol && i > -1 && j > -1)
	{
		dPop[i * nCol + j] = denPop;
		dCase[i * nCol + j] = denCase;
	}
}

void kde(float * caseDen, float * popDen, int nRow, int nCol, float cellSize, float xMin, float yMax, float * xCol, float * yCol, float * pCount, float * cCount, int nHH, float bandwidth)
{
	int gridX = ceil((float) nCol / BLOCKSIZE);
	int gridY = ceil((float) nRow / BLOCKSIZE);

	float blockSizeE = BLOCKSIZE * cellSize;

	int blockBandwidth = ceil(bandwidth / blockSizeE);
	printf("block bandwidth: %d\n", blockBandwidth);

	int dataGridX = gridX + 2 * blockBandwidth;
	int dataGridY = gridY + 2 * blockBandwidth;


	float xMinData = xMin - blockSizeE * blockBandwidth;
	float yMaxData = yMax + blockSizeE * blockBandwidth;

	int rowID, colID, gridID;

	int * nPointsB;
	if(NULL == (nPointsB = (int *) malloc(sizeof(int) * dataGridX * dataGridY)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}
	for(int i = 0; i < dataGridX * dataGridY; i++)
	{
		nPointsB[i] = 0;
	}

	int * dGridID;
	if(NULL == (dGridID = (int *) malloc(sizeof(int) * nHH)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}

	for(int i = 0; i < nHH; i++)
	{
		colID = (int)((xCol[i] - xMinData) / blockSizeE);
		rowID = (int)((yMaxData - yCol[i]) / blockSizeE);
		gridID = rowID * dataGridX + colID;

		if(colID < 0 || colID >= dataGridX || rowID < 0 || rowID >= dataGridY)
		{
			dGridID[i] = -1; 
		}
		else
		{
			nPointsB[gridID] ++;
			dGridID[i] = gridID; 
		}

	}

	int nPointsIn = 0;
	int * startIDB;
	if(NULL == (startIDB = (int *) malloc(sizeof(int) * dataGridX * dataGridY)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}

	for(int i = 0; i < dataGridX * dataGridY; i++)
	{
		startIDB[i] = nPointsIn;
		nPointsIn += nPointsB[i];
	}

	float * xColOrd;
	float * yColOrd;
	float * pCOrd;
	float * cCOrd;

	if(NULL == (xColOrd = (float *) malloc(sizeof(float) * nPointsIn)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (yColOrd = (float *) malloc(sizeof(float) * nPointsIn)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (pCOrd = (float *) malloc(sizeof(float) * nPointsIn)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (cCOrd = (float *) malloc(sizeof(float) * nPointsIn)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}

	for(int i = 0; i < dataGridX * dataGridY; i++)
	{
		nPointsB[i] = startIDB[i];
	}

	for(int i = 0; i < nHH; i++)
	{
		gridID = dGridID[i];
		if(gridID < 0)
			continue;
		xColOrd[nPointsB[gridID]] = xCol[i];
		yColOrd[nPointsB[gridID]] = yCol[i];
		pCOrd[nPointsB[gridID]] = pCount[i];
		cCOrd[nPointsB[gridID]] = cCount[i];
		nPointsB[gridID] ++;
	}

	float * dX;
	float * dY;
	float * dP;
	float * dC;
	int * dPoints;

	float * dPop;
	float * dCase;

	dim3 dimBlock (BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid (gridX, gridY);

	CudaSafeCall(hipMalloc((void **) &dPop, sizeof(float) * nRow * nCol));
	CudaSafeCall(hipMalloc((void **) &dCase, sizeof(float) * nRow * nCol));

	CudaSafeCall(hipMalloc((void **) &dX, sizeof(float) * nPointsIn));
	CudaSafeCall(hipMalloc((void **) &dY, sizeof(float) * nPointsIn));
	CudaSafeCall(hipMalloc((void **) &dP, sizeof(float) * nPointsIn));
	CudaSafeCall(hipMalloc((void **) &dC, sizeof(float) * nPointsIn));
	CudaSafeCall(hipMalloc((void **) &dPoints, sizeof(int) * dataGridX * dataGridY));
	
	CudaSafeCall(hipMemcpy(dX, xColOrd, sizeof(float) * nPointsIn, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dY, yColOrd, sizeof(float) * nPointsIn, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dP, pCOrd, sizeof(float) * nPointsIn, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dC, cCOrd, sizeof(float) * nPointsIn, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(dPoints, nPointsB, sizeof(int) * dataGridX * dataGridY, hipMemcpyHostToDevice));

	CudaCheckError();

	//Kernel Goes here
	kdeKernel<<<dimGrid, dimBlock>>>(dPop, dCase, nRow, nCol, xMin, yMax, cellSize, dX, dY, dP, dC, dPoints, bandwidth * bandwidth, blockBandwidth);


	CudaSafeCall(hipMemcpy(popDen, dPop, sizeof(float) * nRow * nCol, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(caseDen, dCase, sizeof(float) * nRow * nCol, hipMemcpyDeviceToHost));
	hipFree(dPop);
	hipFree(dCase);
	hipFree(dX);
	hipFree(dY);
	hipFree(dP);
	hipFree(dC);
	hipFree(dPoints);

	free(xColOrd);
	free(yColOrd);
	free(pCOrd);
	free(cCOrd);

	free(dGridID);
	free(nPointsB);
	free(startIDB);
}
