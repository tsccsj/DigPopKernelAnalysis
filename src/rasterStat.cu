#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "io.cuh"


//rasterStat inputFileName inputCount inputPCcount xMin yMin xMax yMax cellSize outputFileName
int rasterStat(char * inputFileName, int minRzn, int numRzn, char * inputPCName, float xMin, float yMin, float xMax, float yMax, float cellSize, char * outputFileName, int epsgCode)
{
	FILE ** inputFilesC;
	FILE ** inputFilesP;
	FILE * inputPCcountFile;

//	FILE * outputFile;
	
	int nRow, nCol;

	int * nPop, * nCase;

	char tempFileName[500];


	float * rowCase;
	float * rowPop;

	float * cellCase;
	float * cellPop;
//	float * cellLike;

	int nNA;
	float * mean;
	float * max;
	float * min;
	float * median;
	float * q1;
	float * q3;
	float * meanPop;
	float * range;
	float * iqr;
	int * notNA;
	float * sd;
/*
	float * meanL;
	float * maxL;
	float * minL;
	float * medianL;
	float * q1L;
	float * q3L;
*/	

	if(numRzn < 0 || numRzn > 10000)
	{
		printf("invalid numOfMaps, should be more than 0 and less than 10000\n");
		exit(1);
	}

	if(NULL == (inputFilesC = (FILE **) malloc(sizeof(FILE *) * numRzn)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (inputFilesP = (FILE **) malloc(sizeof(FILE *) * numRzn)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}

	for(int i = 0; i < numRzn; i++)
	{
		sprintf(tempFileName, "%s_rzn%03d", inputFileName, (minRzn + i));
		if(NULL == (inputFilesC[i] = fopen(tempFileName, "rb")))
		{
			printf("ERROR: Can't open input case file: %s.\n", tempFileName);
			exit(1);
		}
		sprintf(tempFileName, "%s_rzn%03d_P", inputFileName, (minRzn + i));
		if(NULL == (inputFilesP[i] = fopen(tempFileName, "rb")))
		{
			printf("ERROR: Can't open input population file: %s.\n", tempFileName);
			exit(1);
		}
	}
	
	if(NULL == (inputPCcountFile = fopen(inputPCName, "r")))
	{
		printf("ERROR: Can't open input population-and-case-count file: %s.\n", tempFileName);
		exit(1);
	}

	if(NULL == (nPop = (int *)malloc(sizeof(int) * numRzn)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (nCase = (int *)malloc(sizeof(int) * numRzn)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}

	for(int i = 0; i < numRzn; i++)
	{
		fscanf(inputPCcountFile, "%s %d %d\n", tempFileName, nPop + i, nCase + i);
	}

	fclose(inputPCcountFile);

	nCol = ceil((xMax - xMin)/cellSize);
	nRow = ceil((yMax - yMin)/cellSize);

	xMax = xMin + cellSize * nCol;
	yMax = yMin + cellSize * nRow;

//	printf("####################\n");
//	printf("nRow: %d\tnCol: %d\n", nRow, nCol);
//	printf("xMax: %f\txMin: %f\nyMax: %f\tyMin: %f\n",xMax,xMin,yMax,yMin);
//	printf("####################\n");


	if(NULL == (rowCase = (float *) malloc (sizeof(float) * numRzn * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (rowPop = (float *) malloc (sizeof(float) * numRzn * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}

	if(NULL == (cellCase = (float *) malloc (sizeof(int) * numRzn)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (cellPop = (float *) malloc (sizeof(int) * numRzn)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
/*
	if(NULL == (cellLike = (float *) malloc (sizeof(int) * numRzn)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
*/
	if(NULL == (notNA = (int *) malloc (sizeof(int) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (mean = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (max = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (min = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (median = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (q1 = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (q3 = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (meanPop = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (range = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (iqr = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (sd = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
/*
	if(NULL == (meanL = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (maxL = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (minL = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (medianL = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (q1L = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (q3L = (float *) malloc (sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
*/	

	float tempF;
	float tempFS;
	for(int i = 0; i < nRow; i++)
	{
		for(int k = 0; k < numRzn; k++)
		{
			if(nCol != fread(rowCase + k * nCol, sizeof(float), nCol, inputFilesC[k]))
			{
				printf("Wrong input file size or nRow, nCol!\n");
				exit(1);
			}
			if(nCol != fread(rowPop + k * nCol, sizeof(float), nCol, inputFilesP[k]))
			{
				printf("Wrong input file size or nRow, nCol!\n");
				exit(1);
			}
		}

		for(int j = 0; j < nCol; j++)
		{
			nNA = 0;
			for(int k = 0; k < numRzn; k++)
			{
				if(rowPop[k * nCol + j] > 0)
				{
					cellCase[nNA] = rowCase[k * nCol + j];
					cellPop[nNA] = rowPop[k * nCol + j];
					//cellLike[nNA] = pow(cellCase[nNA]/cellPop[nNA], cellCase[nNA]) * pow((cellPop[nNA] - cellCase[nNA])/cellPop[nNA], (cellPop[nNA] - cellCase[nNA])) * pow((nCase[k] - cellCase[nNA])/(nPop[k] - cellPop[nNA]), (nCase[k] - cellCase[nNA])) * pow((nPop[k] - cellPop[nNA] - nCase[k] + cellCase[nNA])/(nPop[k] - cellPop[nNA]), (nPop[k] - cellPop[nNA] - nCase[k] + cellCase[nNA]));
					//cellLike[nNA] = cellCase[nNA] * log(cellCase[nNA]/cellPop[nNA]) + (cellPop[nNA] - cellCase[nNA]) * log((cellPop[nNA] - cellCase[nNA])/cellPop[nNA]) + (nCase[k] - cellCase[nNA]) * log((nCase[k] - cellCase[nNA])/(nPop[k] - cellPop[nNA])) + (nPop[k] - cellPop[nNA] - nCase[k] + cellCase[nNA]) * log((nPop[k] - cellPop[nNA] - nCase[k] + cellCase[nNA])/(nPop[k] - cellPop[nNA]));  
					cellCase[nNA] = cellCase[nNA] / cellPop[nNA];

					nNA ++;
				}
			}


			if(0 == nNA)
			{
				mean[i * nCol + j] = -1;
				max[i * nCol + j] = -1;
				min[i * nCol + j] = -1;
				median[i * nCol + j] = -1;
				q1[i * nCol + j] = -1;
				q3[i * nCol + j] = -1;
				meanPop[i * nCol + j] = -1;
				range[i * nCol + j] = -1;
				iqr[i * nCol + j] = -1;
				sd[i * nCol + j] = -1;

/*				meanL[i * nCol + j] = 1;
				maxL[i * nCol + j] = 1;
				minL[i * nCol + j] = 1;
				medianL[i * nCol + j] = 1;
				q1L[i * nCol + j] = 1;
				q3L[i * nCol + j] = 1;
*/
			}
			else
			{
				//Sort Porprotion
				for(int k = 0; k < nNA - 1; k++)
				{
					int maxID = k;
					for(int l = k + 1; l < nNA; l++)
					{
						if(cellCase[maxID] < cellCase[l])
						{
							maxID = l;
						}
					}
					if(maxID != k)
					{
						tempF = cellCase[k];
						cellCase[k] = cellCase[maxID];
						cellCase[maxID] = tempF;
					}
				}

				//Calcualte statistics
				max[i * nCol + j] = cellCase[0];
				min[i * nCol + j] = cellCase[nNA-1];
				if(0==nNA%2)
				{
					median[i * nCol + j] = (cellCase[nNA/2] + cellCase[nNA/2-1]) / 2;
				}
				else
				{
					median[i * nCol + j] = cellCase[nNA/2];
				}	
				q3[i * nCol + j] = cellCase[(int)((float)nNA/4-0.5)];
				q1[i * nCol + j] = cellCase[(int)((float)nNA/4*3-0.5)];
				tempF = 0.0;
				tempFS = 0.0;
				for(int k = 0; k < nNA; k++)
				{
					tempF += cellCase[k];
					tempFS += cellCase[k] * cellCase[k];
				}
				mean[i * nCol + j] = tempF / nNA;
				sd[i * nCol + j] = sqrt(tempFS / nNA - (tempF / nNA) * (tempF / nNA));
				
				tempF = 0.0;
				for(int k = 0; k < nNA; k++)
				{
					tempF += cellPop[k];
				}
				meanPop[i * nCol + j] = tempF / numRzn;
				range[i * nCol + j] = max[i * nCol + j] - min[i * nCol + j];
				iqr[i * nCol + j] = q3[i * nCol + j] - q1[i * nCol + j];
				
/*
				//Sort Likelihood
				for(int k = 0; k < nNA - 1; k++)
				{
					int maxID = k;
					for(int l = k + 1; l < nNA; l++)
					{
						if(cellLike[maxID] < cellLike[l])
						{
							maxID = l;
						}
					}
					if(maxID != k)
					{
						tempF = cellLike[k];
						cellLike[k] = cellLike[maxID];
						cellLike[maxID] = tempF;
					}
				}
				maxL[i * nCol + j] = cellLike[0];
				minL[i * nCol + j] = cellLike[nNA-1];
				if(0==nNA%2)
				{
					medianL[i * nCol + j] = (cellLike[nNA/2] + cellLike[nNA/2-1]) / 2;
				}
				else
				{
					medianL[i * nCol + j] = cellLike[nNA/2];
				}	
				q3L[i * nCol + j] = cellLike[(int)((float)nNA/4-0.5)];
				q1L[i * nCol + j] = cellLike[(int)((float)nNA/4*3-0.5)];
				tempF = 0;
				for(int k = 0; k < nNA; k++)
				{
					tempF += cellLike[k];
				}
				meanL[i * nCol + j] = tempF / nNA;
*/
			}
			notNA[i * nCol + j] = nNA;
		}
	}


	for(int i = 0; i < numRzn; i++)
	{
		fclose(inputFilesC[i]);
		fclose(inputFilesP[i]);
	}


	
	//Write output files
	sprintf(tempFileName, "%s_mean.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, mean, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, mean, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_max.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, max, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, max, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_min.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, min, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, min, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_1q.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, q1, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, q1, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_3q.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, q3, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, q3, nRow, nCol, xMin, yMax, cellSize, epsgCode);
	
	sprintf(tempFileName, "%s_median.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, median, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, median, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_nNA.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridI(outputFile, notNA, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffI(tempFileName, notNA, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_meanPop.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, meanPop, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, meanPop, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_range.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, range, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, range, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_iqr.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, iqr, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, iqr, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	sprintf(tempFileName, "%s_sd.tif", outputFileName);
//	if(NULL == (outputFile = fopen(tempFileName, "w")))
//	{
//		printf("ERROR: Can't open output file%s.\n", tempFileName);
//		exit(1);
//	}
//	writeGridF(outputFile, sd, nRow, nCol, xMin, yMin, cellSize);
//	fclose(outputFile);
	writeGeoTiffF(tempFileName, sd, nRow, nCol, xMin, yMax, cellSize, epsgCode);
/*
	sprintf(tempFileName, "%s_meanLikelihood.asc", outputFileName);
	if(NULL == (outputFile = fopen(tempFileName, "w")))
	{
		printf("ERROR: Can't open output file%s.\n", tempFileName);
		exit(1);
	}
	writeGridF(outputFile, meanL, nRow, nCol, xMin, yMin, cellSize, 1);
	fclose(outputFile);

	sprintf(tempFileName, "%s_maxLikelihood.asc", outputFileName);
	if(NULL == (outputFile = fopen(tempFileName, "w")))
	{
		printf("ERROR: Can't open output file%s.\n", tempFileName);
		exit(1);
	}
	writeGridF(outputFile, maxL, nRow, nCol, xMin, yMin, cellSize, 1);
	fclose(outputFile);

	sprintf(tempFileName, "%s_minLikelihood.asc", outputFileName);
	if(NULL == (outputFile = fopen(tempFileName, "w")))
	{
		printf("ERROR: Can't open output file%s.\n", tempFileName);
		exit(1);
	}
	writeGridF(outputFile, minL, nRow, nCol, xMin, yMin, cellSize, 1);
	fclose(outputFile);

	sprintf(tempFileName, "%s_1qLikelihood.asc", outputFileName);
	if(NULL == (outputFile = fopen(tempFileName, "w")))
	{
		printf("ERROR: Can't open output file%s.\n", tempFileName);
		exit(1);
	}
	writeGridF(outputFile, q1L, nRow, nCol, xMin, yMin, cellSize, 1);
	fclose(outputFile);

	sprintf(tempFileName, "%s_3qLikelihood.asc", outputFileName);
	if(NULL == (outputFile = fopen(tempFileName, "w")))
	{
		printf("ERROR: Can't open output file%s.\n", tempFileName);
		exit(1);
	}
	writeGridF(outputFile, q3L, nRow, nCol, xMin, yMin, cellSize, 1);
	fclose(outputFile);
	
	sprintf(tempFileName, "%s_medianLikelihood.asc", outputFileName);
	if(NULL == (outputFile = fopen(tempFileName, "w")))
	{
		printf("ERROR: Can't open output file%s.\n", tempFileName);
		exit(1);
	}
	writeGridF(outputFile, medianL, nRow, nCol, xMin, yMin, cellSize, 1);
	fclose(outputFile);
*/
	//Clean up resourses
	free(inputFilesC);
	free(inputFilesP);


	free(rowCase);
	free(rowPop);

	free(cellCase);
	free(cellPop);
//	free(cellLike);

	free(notNA);
	free(mean);
	free(max);
	free(min);
	free(median);
	free(q1);
	free(q3);
	free(meanPop);
	free(range);
	free(iqr);
	free(sd);
/*
	free(meanL);
	free(maxL);
	free(minL);
	free(medianL);
	free(q1L);
	free(q3L);
*/
	free(nPop);
	free(nCase);

	//printf("Finished!\n");
	return 0;
}
