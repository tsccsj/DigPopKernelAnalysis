#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "io.cuh"
#include "kde.cuh"
#include <sys/time.h>


//dpbKDE  inputPopFile inputHHFile outputFileName outputCount xMin yMin xMax yMax cellSize bandwidth mapLogic
int dpbKDE(char * inputPopFileName, char * inputHHFileName, char * outputFileName, char * outputCount, float xMin, float yMin, float xMax, float yMax, float cellSize, float bandwidth, char * subPop, char * mapLogic, char * personColName, int epsgCode)
{
	//Uboyt
	FILE * inputPopData;
	FILE * inputHHData;

	char outputRatioGTiffFile[200];
	char outputPopulationFile[200];

	bool hhOnly;	
	
	float * xCol;
	float * yCol;
	
	float * pCount;
	float * cCount;

	int nRow, nCol, nHH, nPop, nCase = 0;
	float * caseDen;
	float * popDen;

	struct timeval time1;
	gettimeofday(&time1, NULL);

	if(strcmp(inputPopFileName, "-HHO") == 0)
	{
		hhOnly = true;
	}
	else
	{
		hhOnly = false;
		if(NULL == (inputPopData = fopen(inputPopFileName, "r")))
		{
			printf("ERROR: Can't open input population file: %s\n", inputPopFileName);
			exit(1);
		}
	}

	if(NULL == (inputHHData = fopen(inputHHFileName, "r")))
	{
		printf("ERROR: Can't open input household file: %s\n", inputHHFileName);
		exit(1);
	}

	sprintf(outputPopulationFile, "%s_P", outputFileName);
	sprintf(outputRatioGTiffFile, "%s.tif", outputFileName);

//Cells
	nCol = ceil((xMax - xMin)/cellSize);
	nRow = ceil((yMax - yMin)/cellSize);

	xMax = xMin + cellSize * nCol;
	yMax = yMin + cellSize * nRow;

	//printf("####################\n");
	//printf("nRow: %d\tnCol: %d\n", nRow, nCol);
	//printf("xMax: %f\txMin: %f\nyMax: %f\tyMin: %f\n",xMax,xMin,yMax,yMin);
	//printf("####################\n");

//Points

	nHH = getHHNum(inputHHData);
	if(NULL == (xCol = (float *)malloc(sizeof(float) * nHH)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (yCol = (float *)malloc(sizeof(float) * nHH)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (pCount = (float *)malloc(sizeof(float) * nHH)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}
	if(NULL == (cCount = (float *)malloc(sizeof(float) * nHH)))
	{
		printf("ERROR: Out of memory in line %d!\n", __LINE__);
		exit(1);
	}

	bool succeed;

	if(hhOnly)
	{
		if(strcmp(subPop, "*") == 0)
		{
			succeed = readPointsH(inputHHData, nHH, nCase, xCol, yCol, pCount, cCount, mapLogic);
		}
		else
		{
			succeed = readPointsHInSubPop(inputHHData, nHH, nCase, xCol, yCol, pCount, cCount, subPop, mapLogic);
		}

		if(!succeed)
		{
			printf("File involved: %s\n", inputHHFileName);
			exit(1);
		}
		//printf("num of household: %d\nnum of positive household: %d\n", nHH, nCase);
		//printf("####################\n");
	}
	else
	{
		if(strcmp(subPop, "*") == 0)
		{
			succeed = readPointsP(inputHHData, inputPopData, nHH, nPop, nCase, xCol, yCol, pCount, cCount, mapLogic, personColName);
		}
		else
		{
			succeed = readPointsPInSubPop(inputHHData, inputPopData, nHH, nPop, nCase, xCol, yCol, pCount, cCount, subPop, mapLogic, personColName);
		}
	
		if(!succeed)
		{
			printf("File involved: %s and\\or %s\n", inputHHFileName, inputPopFileName);
			exit(1);
		}
		//printf("num of household: %d\nnum of population: %d\nnum of case: %d\n", nHH, nPop, nCase);
		//printf("####################\n");
	}

	fclose(inputHHData);
	if(!hhOnly)
	{
		fclose(inputPopData);
	}

	struct timeval time2;
	gettimeofday(&time2, NULL);
	//KDE

	if(NULL == (caseDen = (float *) malloc(sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}
	
	if(NULL == (popDen = (float *) malloc(sizeof(float) * nRow * nCol)))
	{
		printf("ERROR: Out of memory in %d!\n", __LINE__);
		exit(1);
	}

	for(int i = 0; i < nRow * nCol; i++)
	{
		caseDen[i] = 0;
		popDen[i] = 0;
	}

	int x, y;
	if(bandwidth > 1)
	{
		kde(caseDen, popDen, nRow, nCol, cellSize, xMin, yMax, xCol, yCol, pCount, cCount, nHH, bandwidth);

		//filter out non-value areas
		bool * hasValue;
		if(NULL == (hasValue = (bool *) malloc (sizeof(float) * nRow * nCol)))
		{
			printf("ERROR: Out of memory in %d!\n", __LINE__);
			exit(1);
		}
		for(int i = 0; i < nCol * nRow; i++)
		{
			hasValue[i] = false;
		}

		for(int i = 0; i < nHH; i++)
		{
			x = (xCol[i] - xMin) / cellSize;
			y = (yMax - yCol[i]) / cellSize;

			//if(x < 0 || x >= nCol || y < 0 || y >= nRow)
			//{
			//	printf("%d\t%f\t%f\n", i, xCol[i], yCol[i]);
			//}
		
			if(x > -1 && x < nCol && y > -1 && y < nRow && !hasValue[y * nCol + x])
				hasValue[y * nCol + x] = true;		
		}

		for(int i = 0; i < nRow * nCol; i++)
		{
			if(!hasValue[i])
			{
				caseDen[i] = 0;
				popDen[i] = 0;
			}
		}

		free(hasValue);
	}
	else
	{
		for(int i = 0; i < nHH; i++)
		{
			x = (xCol[i] - xMin) / cellSize;
			y = (yMax - yCol[i]) / cellSize;

			if(x >= 0 && x < nCol && y >= 0 && y < nRow)
			{

				caseDen[y * nCol + x] += cCount[i];
				popDen[y * nCol + x] += pCount[i];
			}
		}
	}

	struct timeval time3;
	gettimeofday(&time3, NULL);

	printf("Input time:\t%lfms\n", ((&time2)->tv_sec - (&time1)->tv_sec) * 1000 + (double)((&time2)->tv_usec - (&time1)->tv_usec) / 1000);
	printf("KDE time:\t%lfms\n", ((&time3)->tv_sec - (&time2)->tv_sec) * 1000 + (double)((&time3)->tv_usec - (&time2)->tv_usec) / 1000);

	//Write outputFile
	FILE * outputFile;
	if(NULL == (outputFile = fopen(outputFileName, "wb")))
	{
		printf("ERROR: Can't open output file");
		exit(1);
	}
	fwrite(caseDen, sizeof(float), nRow * nCol, outputFile);
	fclose(outputFile);

//Generate ascii grid
/*
	if(NULL == (outputFile = fopen(outputRatioGTiffFile, "w")))
	{
		printf("ERROR: Can't open output ratio file: %s\n", outputRatioGTiffFile);
		exit(1);
	}
	writeGridRatio(outputFile, caseDen, popDen, nRow, nCol, xMin, yMin, cellSize);
	fclose(outputFile);
*/
//Generate GeoTiff
	writeGeoTiffRatio(outputRatioGTiffFile, caseDen, popDen, nRow, nCol, xMin, yMax, cellSize, epsgCode);

	if(NULL == (outputFile = fopen(outputPopulationFile, "wb")))
	{
		printf("ERROR: Can't open output population file: %s\n", outputPopulationFile);
		exit(1);
	}
	fwrite(popDen, sizeof(float), nRow * nCol, outputFile);
	fclose(outputFile);

// This part is used to be used to calcuate the likelihood, but is no longer used
	if(NULL == (outputFile = fopen(outputCount, "a")))
	{
		printf("ERROR: Can't open output population and count file: %s\n", outputCount);
		exit(1);
	}
	if(hhOnly)
	{
		fprintf(outputFile, "%s %d %d\n", outputFileName, nHH, nCase);
	}
	else
	{
		fprintf(outputFile, "%s %d %d\n", outputFileName, nPop, nCase);
	}
	fclose(outputFile);
	
	//free
	free(xCol);
	free(yCol);
	free(pCount);
	free(cCount);

	free(caseDen);
	free(popDen);

	//printf("Finished!\n");
	return 0;
}



